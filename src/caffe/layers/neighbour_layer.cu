#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <iostream>
#include <stdio.h>

#include "caffe/blob.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/neighbour_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void NeighbourLayerForward(const int n_threads, const int count,
	const int channels, const int height, const int width,
    const Dtype* in1, const Dtype* in2, Dtype* out1, Dtype* out2, Dtype temp1, Dtype temp2) {
  CUDA_KERNEL_LOOP(index, count) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;

	const Dtype* bot_slice1 = in1 + (n * channels + c) * height * width;
	const Dtype* bot_slice2 = in2 + (n * channels + c) * height * width;
	Dtype value_f1 = bot_slice1[h*width+w];
	Dtype value_f2 = bot_slice2[h*width+w];	

    for (int y = -2; y < 3; ++y){
		for (int x = -2; x < 3; ++x){
			int test1 = h+y;
			int test2 = w+x;

			if (test1 < 0 || test2 < 0 || test1 >= height || test2 >= width){
				temp1 = 0;
				temp2 = 0;
			}
			else{
				temp1 = bot_slice1[(h+y)*width+w+x];
				temp2 = bot_slice2[(h+y)*width+w+x];				
				
			}
			int index_out = ((n * channels + c) * (5*height) + (5*h) + y + 2) * (5*width) + (5*w) + x + 2;
			out1[index_out] = (value_f1-temp2);
			out2[index_out] = (value_f2-temp1);
		}
	}
  }
}

template <typename Dtype>
void NeighbourLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_data2 = bottom[1]->gpu_data();

  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* top_data2 = top[1]->mutable_gpu_data();
  int count = top[0]->count();
  int count_bottom = bottom[0]->count();
  Dtype temp1 = 0;
  Dtype temp2 = 0;
  // NOLINT_NEXT_LINE(whitespace/operators)
  NeighbourLayerForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, count_bottom, channels_i,
      height_i, width_i, bottom_data, bottom_data2, top_data, top_data2, temp1, temp2);
  CUDA_POST_KERNEL_CHECK;
}	
	
template <typename Dtype>
__global__ void NeighbourLayerBackward(const int n_threads, const int count,
	const int channels, const int height, const int width,
    const Dtype* const in1, const Dtype* const in2, Dtype* const out1, Dtype temp1, Dtype temp2, int other) {
  CUDA_KERNEL_LOOP(index, count) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
	
    //Dtype min_val1 = in1[((n * channels + c) * (5*height) + (5*h)) * (5*width) + (5*w)];
    //Dtype min_val2 = in2[((n * channels + c) * (5*height) + (5*h)) * (5*width) + (5*w)];	

    // Vers. 1.0
   /* for (int y = 0; y < 5; ++y){
		for (int x = 0; x < 5; ++x){
			temp1 = in1[((n * channels + c) * (5*height) + (5*h+y)) * (5*width) + (5*w+x)];
			temp2 = in2[((n * channels + c) * (5*height) + (5*h+y)) * (5*width) + (5*w+x)];
			
			if (temp1 < min_val1){
				min_val1 = temp1;
			}
			if (temp2 < min_val2){
				min_val2 = temp2;
			}
		}
	} */ 
	
	int count = 0;
	int index_o = ((n * channels + c) * height + h) * width + w;
	
	// Vers 2.0
		for (int y = 0; y < 5; ++y){
			for (int x = 0; x < 5; ++x){
				count += 1;
				out1[index_o] += in1[((n * channels + c) * (5*height) + (5*h) + y) * 5*width + (5*w) + x];
				if ((x-2)+w >= width || (y-2)+h >= height || (x-2)+w < 0 || (y-2)+h < 0){
					continue;
				}
				else{
					int off_y = 2-(y-2);
					int off_x = 2-(x-2);
					out1[index_o] += -1*in2[((n * channels + c) * (5*height) + 5*((y-2)+h)+off_y) * (5*width) + 5*((x-2)+w) + off_x];
					count += 1;
				}
			}
		} 
		
	out1[index_o] /= count;
	/*if (other == 1){
	out1[index_o] = min_val1;
	}
	else{
	out1[index_o] = min_val2;
	}*/
  }
}

template <typename Dtype>
void NeighbourLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  for (int i = 0; i < bottom.size(); ++i){
	  const int other = (i == 0) ? 1 : 0;
	  
	  const Dtype* top_diff = top[i]->gpu_diff();
	  const Dtype* top_diff2 = top[other]->gpu_diff();
	  Dtype* bottom_data = bottom[i]->mutable_gpu_diff();
	  
	  const int count = top[i]->count();
	  const int bottom_count = bottom[i]->count();
	  Dtype temp1 = 0;
	  Dtype temp2 = 0;

	  // NOLINT_NEXT_LINE(whitespace/operators)
	  NeighbourLayerBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		  count, bottom_count, channels_i,
		  height_i, width_i, top_diff, top_diff2, bottom_data, temp1, temp2, other);
	  CUDA_POST_KERNEL_CHECK;
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(NeighbourLayer);

} // namespace caffe

